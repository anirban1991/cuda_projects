#include <stdio.h>
#include <hip/hip_runtime.h>



__global__
void arrayAddKernel(float* A, float* B, float *C, int n, size_t pitch)
{
    int tidx = threadIdx.x * blockDim.x + blockIdx.x;
    int tidy = threadIdx.y * blockDim.y + blockIdx.y;


    if ((tidx<n) && (tidy <n))
    {
        float *row_a = (float *)((char*)A + tidy * pitch);
        float *row_b = (float *)((char*)B + tidy * pitch);
        float *row_c = (float *)((char*)C + tidy * pitch);
        row_c[tidx] =  row_a[tidx] + row_b[tidx];
        printf("A value %.3f, %d, %d \n", row_a[tidx], tidx, tidy);
        printf("B value %.3f, %d, %d \n", row_b[tidx], tidx, tidy);
        printf("C value %.3f, %d, %d \n", row_c[tidx], tidx, tidy);

    }


}

void array_add(float* A, float* B, float* C, int n)
{
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;
    size_t pitch;

    hipMallocPitch((void **)&d_A, &pitch, size,n);
    hipMemcpy2D(d_A, pitch, A, size, size, n, hipMemcpyHostToDevice);
    hipMallocPitch((void **)&d_B, &pitch, size,n);
    hipMemcpy2D(d_B, pitch, B, size, size, n, hipMemcpyHostToDevice);
    hipMallocPitch((void **)&d_C, &pitch, size,n);
    printf("hello0\n");
    // Kernel Code
    dim3 grid(16,16);
    dim3 block(16,1);

    arrayAddKernel<<< grid, block >>> (d_A, d_B, d_C, n, pitch );
    hipMemcpy2D(C, pitch, d_C, size, size, n, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);


}

int main()
{
    int dim = 3;
    float h_A[3][3] = {{2.5,3.5,5.0},{1.5,3.5,1.0}, {0.0,0.0,0.0}};
    float h_B[3][3] = {{1.5,3.5,0.0},{2.5,3.0,1.5}, {0.0,0.0,0.0}};
    float h_C[3][3];



    array_add((float *)h_A, (float *)h_B, (float *)h_C, dim);

    // NOT getting any out put here 

    for (int i= 0; i<dim ;i++)
    { for (int j = 0 ; j< dim; j++)
    {
        printf("%.f ", h_C[i][j]);
    }
    }

}