// Device code
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
 int i = blockDim.x * blockIdx.x + threadIdx.x;
 if (i < N)
 C[i] = A[i] + B[i];
}
// Host code
int main()
{
 int N = 5;
 size_t size = N * sizeof(float);
 // Allocate input vectors h_A and h_B in host memory
//  float* h_A = (float*)malloc(size);
//  float* h_B = (float*)malloc(size);
 // Initialize input vectors
float h_A[] = {1.5,2.0,4.1,5.2,6.0};
float h_B[] = {1.5, 1.5, 3.0, 3.0, 4.0};
float h_C[5];
 // Allocate vectors in device memory
 float* d_A;
 hipMalloc(&d_A, size);
 float* d_B;
 hipMalloc(&d_B, size);
 float* d_C;
 hipMalloc(&d_C, size);
 // Copy vectors from host memory to device memory
 hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
 hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
 // Invoke kernel

 int threadsPerBlock = 256;
 int blocksPerGrid =
 (N + threadsPerBlock - 1) / threadsPerBlock;
 VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
 // Copy result from device memory to host memory
 // h_C contains the result in host memory
 hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
 // Free device memory
 hipFree(d_A);
 hipFree(d_B);
 hipFree(d_C);
 // Free host memory
}
